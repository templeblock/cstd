
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_pooling_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Forward(GPUContext* context, const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->data<Context>();
  Dtype* top_data = top[0]->mutable_data<Context>();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Backward(GPUContext* context, const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!top[0]->propagate_down_) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->data<Context>();
  const Dtype* bottom_data = bottom[0]->data<Context>();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnPoolingBackward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data, top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNPoolingLayer);

}  // namespace
#endif
